#include "hip/hip_runtime.h"
/**
* @file    3D Optical flow using NVIDIA CUDA
* @author  Institute for Photon Science and Synchrotron Radiation, Karlsruhe Institute of Technology
*
* @date    2015-2018
* @version 0.5.0
*
*
* @section LICENSE
*
* This program is copyrighted by the author and Institute for Photon Science and Synchrotron Radiation,
* Karlsruhe Institute of Technology, Karlsruhe, Germany;
*
*
*/

#include <>

#define __HIPCC__
#include <math_functions.h>

#include "src/data_types/data_structs.h"

#define IIND(X, Y, Z) (((Z) * input_size.height + (Y)) * (input_size.pitch / sizeof(float)) + (X)) 
#define OIND(X, Y, Z) (((Z) * output_size.height + (Y)) * (output_size.pitch / sizeof(float)) + (X)) 

__constant__ DataSize4 input_size;
__constant__ DataSize4 output_size;

extern "C" __global__ void resample_x_p_3d(
  const float* input,
        float* output,
        size_t out_width,
        size_t out_height,
        size_t out_depth,
        size_t in_width)
{
  dim3 globalID(blockDim.x * blockIdx.x + threadIdx.x,
                blockDim.y * blockIdx.y + threadIdx.y,
                blockDim.z * blockIdx.z + threadIdx.z);

  if (globalID.x < out_width &&  globalID.y < out_height && globalID.z < out_depth) {
    float delta = in_width / static_cast<float>(out_width);
    float normalization = out_width / static_cast<float>(in_width);

    float left_f = globalID.x * delta;
    float right_f = (globalID.x + 1) * delta;

    int left_i = static_cast<int>(floor(left_f));
    int right_i = min(in_width, static_cast<size_t>(ceil(right_f)));

    float value = 0.f;
    
    for (int j = 0; j < (right_i - left_i); j++) {
      float frac = 1.f;

      /* left boundary */
      if (j == 0) {
        frac = static_cast<float>(left_i + 1) - left_f;
      }
      /* right boundary */
      if (j == (right_i - left_i) - 1) {
        frac = right_f - static_cast<float>(left_i + j);
      }
      /* if the left and right boundaries are in the same cell */
      if ((right_i - left_i) == 1) {
        frac = delta;
      }
      value += input[IIND(left_i + j, globalID.y, globalID.z)] * frac;
    }
    output[OIND(globalID.x, globalID.y, globalID.z)] = value * normalization;
  }
}

extern "C" __global__ void resample_y_p_3d(
  const float* input,
        float* output,
        size_t out_width,
        size_t out_height,
        size_t out_depth,
        size_t in_height)
{
  dim3 globalID(blockDim.x * blockIdx.x + threadIdx.x,
                blockDim.y * blockIdx.y + threadIdx.y,
                blockDim.z * blockIdx.z + threadIdx.z);

  if (globalID.x < out_width &&  globalID.y < out_height && globalID.z < out_depth) {
    float delta = in_height / static_cast<float>(out_height);
    float normalization = out_height / static_cast<float>(in_height);

    float left_f = globalID.y * delta;
    float right_f = (globalID.y + 1) * delta;

    int left_i = static_cast<int>(floor(left_f));
    int right_i = min(in_height, static_cast<size_t>(ceil(right_f)));

    float value = 0.f;
    
    for (int j = 0; j < (right_i - left_i); j++) {
      float frac = 1.f;

      /* left boundary */
      if (j == 0) {
        frac = static_cast<float>(left_i + 1) - left_f;
      }
      /* right boundary */
      if (j == (right_i - left_i) - 1) {
        frac = right_f - static_cast<float>(left_i + j);
      }
      /* if the left and right boundaries are in the same cell */
      if ((right_i - left_i) == 1) {
        frac = delta;
      }
      value += input[IIND(globalID.x, left_i + j, globalID.z)] * frac;
    }
    output[OIND(globalID.x, globalID.y, globalID.z)] = value * normalization;
  }
}

extern "C" __global__ void resample_z_p_3d(
  const float* input,
        float* output,
        size_t out_width,
        size_t out_height,
        size_t out_depth,
        size_t in_depth)
{
  dim3 globalID(blockDim.x * blockIdx.x + threadIdx.x,
                blockDim.y * blockIdx.y + threadIdx.y,
                blockDim.z * blockIdx.z + threadIdx.z);

  if (globalID.x < out_width &&  globalID.y < out_height && globalID.z < out_depth) {
    float delta = in_depth / static_cast<float>(out_depth);
    float normalization = out_depth / static_cast<float>(in_depth);

    float left_f = globalID.z * delta;
    float right_f = (globalID.z + 1) * delta;

    int left_i = static_cast<int>(floor(left_f));
    int right_i = min(in_depth, static_cast<size_t>(ceil(right_f)));

    float value = 0.f;
    
    for (int j = 0; j < (right_i - left_i); j++) {
      float frac = 1.f;

      /* left boundary */
      if (j == 0) {
        frac = static_cast<float>(left_i + 1) - left_f;
      }
      /* right boundary */
      if (j == (right_i - left_i) - 1) {
        frac = right_f - static_cast<float>(left_i + j);
      }
      /* if the left and right boundaries are in the same cell */
      if ((right_i - left_i) == 1) {
        frac = delta;
      }
      value += input[IIND(globalID.x, globalID.y, left_i + j)] * frac;
    }
    output[OIND(globalID.x, globalID.y, globalID.z)] = value * normalization;
  }
}